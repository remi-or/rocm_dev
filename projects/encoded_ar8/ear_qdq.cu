#include "hip/hip_runtime.h"
#include "./ear_core.cu"

__device__ void warpwiseQuantize(
    const half2* __restrict__ xHalf,
    fp8x2* __restrict__ xQuantized,
    float2* __restrict__ xQParams
) {
    // Each thread loads ELEMS_PER_THREAD elements (so half of that in packed elements)
    half2 regsHalf[PK_ELEMS_PER_THREAD];
    #pragma unroll
    for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) { regsHalf[i] = xHalf[i]; }

    // Convert to fp32
    float2 regsFloat[PK_ELEMS_PER_THREAD];
    #pragma unroll
    for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) { regsFloat[i] = __half22float2(regsHalf[i]); }

    // Quantize and store
    quantizeAndStore(regsFloat, xQuantized, xQParams);
}


__device__ void warpwiseDequantize(
    const fp8x2* xQuantized,
    const float2* xQParams,
    half2* output
) {
    // Each thread loads ELEMS_PER_THREAD elements (so half of that in packed elements)
    fp8x2 xRegs[PK_ELEMS_PER_THREAD];
    #pragma unroll
    for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) { xRegs[i] = xQuantized[i]; }

    // Also load the scales for x and y
    const float2 qParams = xQParams[0];

    // Dequantize to fp32, scale and convert to fp16
    half2 regsHalf[PK_ELEMS_PER_THREAD];

    #pragma unroll
    for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) {
        float2 tmp = dequantizeFp8x2(xRegs[i], qParams);
        regsHalf[i] = __float22half2_rn(tmp);
    }

    // Store back
    #pragma unroll
    for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) {
        output[i] = regsHalf[i];
    }
}

__device__ void warpwiseInplaceDQAQ( // Dequantize, Accumulate, Quantize in place
    fp8x2* xQuantized,
    const fp8x2* yQuantized,
    float2* xQParams,
    const float2* yQParams
) {
    // Each thread loads ELEMS_PER_THREAD elements (so half of that in packed elements)
    fp8x2 xRegs[PK_ELEMS_PER_THREAD];
    #pragma unroll
    for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) { xRegs[i] = xQuantized[i]; }

    fp8x2 yRegs[PK_ELEMS_PER_THREAD];
    #pragma unroll
    for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) { yRegs[i] = yQuantized[i]; }

    // Also load the scales for x and y
    const float2 xQp = xQParams[0];
    const float2 yQp = yQParams[0];

    // Dequantize to fp32
    float2 regsFp[PK_ELEMS_PER_THREAD];
#pragma unroll
    for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) {
        regsFp[i] = dequantizeFp8x2(xRegs[i], xQp) + dequantizeFp8x2(yRegs[i], yQp);
    }

    // Store back as quantized values
    quantizeAndStore(regsFp, xQuantized, xQParams);
}

// 0 1 2 3
// 01 01 23 23
// 0123 0123 0123 0123

// 0 1 2 3
// 01 12 23 30
// 012 123 230 301
// 0123 1230 2301 3012

// 0 1 2 3 4 5 6 7
// 01 01 23 23 45 45 67 67
// 0123 0123 2345 2345 4567 4567 6701 6701
// 01234567 01234567 01234567 01234567 01234567 01234567 01234567 01234567


// __device__ void warpwiseDequantizeAccumulate(
//     fp8x2* xQuantized,
//     fp8x2* yQuantized,
//     __half2* xQParams,
//     const __half2* yQParams,
//     half2* output
// ) {
//     const int threadId = threadIdx.x;
//     const int warpId = threadIdx.x % WARPSIZE;

//     // Relocate in sources
//     xQuantized += threadId * PK_ELEMS_PER_THREAD;
//     yQuantized += threadId * PK_ELEMS_PER_THREAD;

//     // Each thread loads ELEMS_PER_THREAD elements (so half of that in packed elements)
//     fp8x2 xRegs[PK_ELEMS_PER_THREAD];
//     #pragma unroll
//     for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) { xRegs[i] = xQuantized[i]; }

//     fp8x2 yRegs[PK_ELEMS_PER_THREAD];
//     #pragma unroll
//     for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) { yRegs[i] = yQuantized[i]; }

//     // Also load the scales for x and y
//     const float scale = xScales[warpId] * yScales[warpId];

//     // Dequantize to fp32, scale and convert to fp16
//     half2 xyRegs[PK_ELEMS_PER_THREAD];

//     #pragma unroll
//     for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) {
//         // TODO: try dq and mul
//         float2 xy = dequantizeFp8x2(xRegs[i]) * dequantizeFp8x2(yRegs[i]);
//         xy *= scale;
//         xyRegs[i] = __float22half2_rn(xy);
//     }

//     // Store back
//     #pragma unroll
//     for (int i = 0; i < PK_ELEMS_PER_THREAD; i++) {
//         output[i] = xyRegs[i];
//     }
// }
