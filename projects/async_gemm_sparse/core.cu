#pragma once

#include <hip/hip_runtime.h>
#include <iostream>

// #include <ATen/cuda/HIPContext.h>
// #include <c10/cuda/CUDAGuard.h>
// #include <hip/hip_fp8.h>
// #include <torch/all.h>
// using fp8 = __hip_fp8_storage_t;
// using fp8x8 = __attribute__( (__vector_size__(8 * sizeof(fp8)) )) fp8;
// using fp8x16 = __attribute__( (__vector_size__(16 * sizeof(fp8)) )) fp8;
// using fp8_4x2 = __attribute__( (__vector_size__(2 * sizeof(int)) )) int;
// using fp8_4x4 = __attribute__( (__vector_size__(4 * sizeof(int)) )) int;
// using f32x4 = __attribute__( (__vector_size__(4 * sizeof(float)) )) float;
// using uint8 = unsigned char;
// using uint16 = unsigned short;
// using uint32 = unsigned int;
// using uint64 = unsigned long long;

#define WARPSIZE 64
#define OP_M 8
#define OP_N 16
#define OP_K 64

#define SMEM_BANKS 32
#define E_PER_BANK 4

#define A_PRODUCERS 2
#define B_PRODUCERS 3
#define CONSUMERS 2

#define WARPTILE_M OP_M
#define WARPTILE_N (OP_N * 4)
#define WARPTILE_K OP_K
#define PRODUCED_MASK 257

#define QSIZE 24
#define G_ATOMICS true
#define SPLIT_K 1

#define K_BLOCKS(k) (((k / WARPTILE_K) / SPLIT_K))

int inline __device__ infer_k_blocks(const int &k) {
    if (SPLIT_K == 1) {
        return k / WARPTILE_K;
    } else {
        if (blockIdx.z < SPLIT_K - 1) {
            return K_BLOCKS(k);
        } else {
            return (k / WARPTILE_K) - (SPLIT_K - 1) * K_BLOCKS(k);
        }
    }
}
