#include "hip/hip_runtime.h"
#include "./../common.cuh"
#include "./sparse_k.cu"

#include <hip/hip_fp16.h>

template <typename out_dtype>
void host_tiled_sum_reduce(
    const fp8* __restrict__ A, 
    const fp8* __restrict__ B,
    out_dtype* __restrict__ &D, 
    const int m, 
    const int n, 
    const int k
) {
    float acc;
    const fp8* a;
    const fp8* b;

    // Allocate result tensor
    D = (out_dtype*) malloc(m * n * sizeof(out_dtype));

    // Square loop
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {

            // Setup K-wise loop
            acc = 0;
            a = A + i * k;
            b = B + j * k;
            // K-wise loop
            for (int l = 0; l < k; l++) {
                acc += __hip_cvt_fp8_to_halfraw(a[l], __HIP_E4M3_FNUZ).data * __hip_cvt_fp8_to_halfraw(b[l], __HIP_E4M3_FNUZ).data;
            }
            // Store back
            D[i * n + j] = (out_dtype) acc;
        }
    }
}

#define OUTD float

int main(int argc, char **argv) {
    HIP_CHECK( hipSetDevice(0) );

    assert(argc==4);
    const int m = atoi(argv[1]);
    const int n = atoi(argv[2]);
    const int k = atoi(argv[3]);

    // Host tensors
    fp8 *hA, *hB;
    OUTD* host_ref;
    fp8 elem = __hip_cvt_float_to_fp8(1.0f, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
    random_host_tensor<fp8>(hA, m * k); // full_host_tensor<fp8>(hA, m * k, elem);
    random_host_tensor<fp8>(hB, n * k); // full_host_tensor<fp8>(hB, n * k, 0); // random_host_tensor<fp8>(hB, n * k);
    // for (int i = 16; i < 32; i++) {
    //     hB[i] = __hip_cvt_float_to_fp8(1.0f, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
    // }
    host_tiled_sum_reduce<OUTD>(hA, hB, host_ref, m, n, k);

    // Device tensors
    fp8 *dA, *dB;
    OUTD* dD;
    tensor_h2d<fp8>(hA, dA, m * k);
    tensor_h2d<fp8>(hB, dB, n * k);
    zero_device_tensor<OUTD>(dD, m * n);

    HIP_CHECK( hipDeviceSynchronize() );
    async_gemm(dA, dB, dD, m, n, k);
    HIP_CHECK( hipDeviceSynchronize() );

    // Transfer result and free device tensors
    OUTD* host_result;
    tensor_d2h<OUTD>(dD, host_result, m * n);
    HIP_CHECK( hipDeviceSynchronize() );
    HIP_CHECK(hipFree(dA));
    HIP_CHECK(hipFree(dB));
    HIP_CHECK(hipFree(dD));

    // Compare the two host tensors
    float delta;
    float sum_delta = 0.0f;
    float max_delta = 0.0f;
    for (int k = 0; k < m * n; k++) {
        delta = abs((float) host_result[k] - (float) host_ref[k]);
        sum_delta += delta;
        max_delta = (delta > max_delta) ? delta : max_delta;
        // std::cout << host_ref[k] << ":" << host_result[k] << ", ";
    }
    std::cout << "{\"max_delta\": " << max_delta << ", \"total_delta\": " << sum_delta << "}";

    // Free host-side tensors
    free(hA);
    free(hB);
    free(host_ref);
    free(host_result);
}
