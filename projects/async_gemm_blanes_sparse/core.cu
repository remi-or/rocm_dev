#pragma once

#include <hip/hip_runtime.h>
#include <iostream>

// #include <ATen/cuda/HIPContext.h>
// #include <c10/cuda/CUDAGuard.h>
// #include <hip/hip_fp8.h>
// #include <torch/all.h>
// using fp8 = __hip_fp8_storage_t;
// using fp8x8 = __attribute__( (__vector_size__(8 * sizeof(fp8)) )) fp8;
// using fp8x16 = __attribute__( (__vector_size__(16 * sizeof(fp8)) )) fp8;
// using fp8_4x2 = __attribute__( (__vector_size__(2 * sizeof(int)) )) int;
// using fp8_4x4 = __attribute__( (__vector_size__(4 * sizeof(int)) )) int;
// using f32x4 = __attribute__( (__vector_size__(4 * sizeof(float)) )) float;
// using uint8 = unsigned char;
// using uint16 = unsigned short;
// using uint32 = unsigned int;
// using uint64 = unsigned long long;

#define WARPSIZE 64
#define OP_M 8
#define OP_N 16
#define OP_K 64
#define E_P_BANK 4
#define NB_BANKS 32

#define WARPTILE_M OP_M
#define WARPTILE_N (OP_N * B_LANES)
#define WARPTILE_K OP_K
#define PRODUCED_MASK 257
#define G_ATOMICS true

#define A_PRODUCERS 3
#define B_PRODUCERS 3
#define CONSUMERS 4

#define B_LANES 3

#define TIED_CONSUMER false
#define QSIZE 12
#define SPLIT_K 1


// A - 35 57 37 63
// B0 - 202 82 214 259
// B1 - 0 47 0 0 
// B2 - 0 515 472 512
// C - 0 0 0 0 

#define ELEMS_PER_THREADS ((WARPTILE_M * WARPTILE_N) / WARPSIZE)
#define THREADS_PER_ROW (WARPTILE_N / ELEMS_PER_THREADS)


#define K_BLOCKS(k) (((k / WARPTILE_K) / SPLIT_K))

int inline __device__ infer_k_blocks(const int &k) {
    if (SPLIT_K == 1) {
        return k / WARPTILE_K;
    } else {
        if (blockIdx.z < SPLIT_K - 1) {
            return K_BLOCKS(k);
        } else {
            return (k / WARPTILE_K) - (SPLIT_K - 1) * K_BLOCKS(k);
        }
    }
}
