#include "hip/hip_runtime.h"
#include "./core.cu"

void inline __device__ consumer_smem_to_reg8(fp8* buffer, fp8x8 &reg) 
{
    // 32 bits load from the current bank
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        reg[i] = buffer[i];
    }
    // 32 bits load from the same bank, hopefully extension of the first load
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        reg[4 + i] = buffer[i + 32*E_P_BANK];
    }
}

void inline __device__ consumer_smem_to_reg16(fp8* buffer, fp8x16 &reg) 
{
    #pragma unroll
    for (int i = 0; i < 4; i++) { reg[i     ] = buffer[i                   ]; }
    #pragma unroll
    for (int i = 0; i < 4; i++) { reg[i +  4] = buffer[i +     32*E_P_BANK]; }
    #pragma unroll
    for (int i = 0; i < 4; i++) { reg[i +  8] = buffer[i + 2 * 32*E_P_BANK]; }
    #pragma unroll
    for (int i = 0; i < 4; i++) { reg[i + 12] = buffer[i + 3 * 32*E_P_BANK]; }
}

template <typename T>
void __device__ _tsr_consumer(
    fp8* A_buffer,
    fp8* B_buffer,
    T* D,
    uint8* queue,
    int &index,
    uint8 &p_state,
    int &role_id,
    const int n,
    const int dropped_cols,
    const int k,
    const int k_blocks
);

template <>
void __device__ _tsr_consumer(
    fp8* A_buffer,
    fp8* B_buffer,
    float* D,
    uint8* queue,
    int &index,
    uint8 &p_state,
    int &role_id,
    const int n,
    const int dropped_cols,
    const int k,
    const int k_blocks
) {
    // Compute thread position
    const int thread_id = threadIdx.x % WARPSIZE;
    A_buffer += (thread_id / 2) * E_P_BANK + (threadIdx.x % 2) * 32*E_P_BANK * 2;
    B_buffer += (thread_id % 32) * 4 + (thread_id / 32) * 32*E_P_BANK * 4;
    const int sparsity_indices = (threadIdx.x % 2) ? 0x0000EEEE : 0x00004444;

    // Declare input registers
    fp8x8 reg_A[OPS];
    fp8x16 reg_B[OPS];

    // Initialize output registers
    f32x4 reg_D[B_LANES];
    #pragma unroll
    for (int i = 0; i < (B_LANES); i++) {
        reg_D[i][0] = 0.0f; reg_D[i][1] = 0.0f; reg_D[i][2] = 0.0f; reg_D[i][3] = 0.0f;
    }

    // K-wise loop
    fp8 *A_offs_buff, *B_offs_buff;
    int b = role_id;

    while (b < k_blocks) {

        // Account for cyclic queue
        index -= (index >= QSIZE) ? QSIZE : 0;
        A_offs_buff = A_buffer + index * (WARPTILE_M * WARPTILE_K);
        B_offs_buff = B_buffer + index * (WARPTILE_N * WARPTILE_K);

        // Wait for A buffer to be filled
        while (queue[2 * B_LANES * index] != p_state) {
            asm volatile("s_sleep 0");
        }
        // Load A buffer
        #pragma unroll
        for (int op = 0; op < OPS; op++) {
            consumer_smem_to_reg8(A_offs_buff + (op * OP_M * OP_K), reg_A[op]);
        }
        // Mark A buffer as consumed
        queue[2 * B_LANES * index] = p_state + 32;

        // Go through each lanes
        #pragma unroll
        for (int lane = 0; lane < B_LANES; lane++) {

            // Wait for B buffer to be filled
            while (queue[2 * (B_LANES * index + lane) + 1] != p_state) {
                asm volatile("s_sleep 0");
            }
            // Load B buffer
            #pragma unroll
            for (int op = 0; op < OPS; op++) {
                consumer_smem_to_reg16(B_offs_buff + (lane * OP_N * WARPTILE_K) + (op * OP_N * OP_K), reg_B[op]);
            }
            // Mark B buffer as consumed
            queue[2 * (B_LANES * index + lane) + 1] = p_state + 32;

            // Consume registers
            #pragma unroll
            for (int op = 0; op < OPS; op++) {
                reg_D[lane] = __builtin_amdgcn_smfmac_f32_16x16x64_fp8_fp8(
                    reinterpret_cast<fp8_4x2>(reg_A[op]),
                    reinterpret_cast<fp8_4x4>(reg_B[op]),
                    reg_D[lane], 
                    sparsity_indices, // src2
                    7, 1 // cbsz, abid
                );
            }
        }

        // Update index
        index += CONSUMERS;
        p_state = (index >= QSIZE) ? p_state + 64 : p_state;
        b += CONSUMERS;
    }

    // Bring warps back in order
    role_id = b - k_blocks;

    // Fuse complementary registers
    int out_n = 2 * ((thread_id % 16) / 2);
    bool kept;
    #pragma unroll
    for (int i = 0; i < B_LANES; i++) {
        kept = (out_n + i * OP_N) >= dropped_cols;
        reg_D[i][0] = (kept) * (reg_D[i][0] + reg_D[i][1]);
        reg_D[i][1] = (kept) * (reg_D[i][2] + reg_D[i][3]);
    }

    // Relocate on D
    int out_m = (thread_id / 16) * 2 + (thread_id % 2);
    D += (out_m * n + out_n);

    // Out lane by lane
    int id_to_swap = 1 - threadIdx.x % 2;
    int src_lane = thread_id + 1 - 2 * (thread_id % 2);
    __half2 x;
    for (int i = 0; i < B_LANES; i++) {

        // Swap registers across the warp
        reg_D[i][id_to_swap] = __shfl(
            reg_D[i][id_to_swap], 
            src_lane
        );
        
        atomicAdd(&D[0 + i*OP_N], reg_D[i][0]);
        atomicAdd(&D[1 + i*OP_N], reg_D[i][1]);
    }
}

template <>
void __device__ _tsr_consumer(
    fp8* A_buffer,
    fp8* B_buffer,
    half* D,
    uint8* queue,
    int &index,
    uint8 &p_state,
    int &role_id,
    const int n,
    const int dropped_cols,
    const int k,
    const int k_blocks
) {
    // Compute thread position
    const int thread_id = threadIdx.x % WARPSIZE;
    A_buffer += (thread_id / 2) * E_P_BANK + (threadIdx.x % 2) * 32*E_P_BANK * 2;
    B_buffer += (thread_id % 32) * 4 + (thread_id / 32) * 32*E_P_BANK * 4;
    const int sparsity_indices = (threadIdx.x % 2) ? 0x0000EEEE : 0x00004444;

    // Declare input registers
    fp8x8 reg_A[OPS];
    fp8x16 reg_B[OPS];

    // Initialize output registers
    f32x4 reg_D[B_LANES];
    #pragma unroll
    for (int i = 0; i < (B_LANES); i++) {
        reg_D[i][0] = 0.0f; reg_D[i][1] = 0.0f; reg_D[i][2] = 0.0f; reg_D[i][3] = 0.0f;
    }

    // K-wise loop
    fp8 *A_offs_buff, *B_offs_buff;
    int b = role_id;

    while (b < k_blocks) {

        // Account for cyclic queue
        index -= (index >= QSIZE) ? QSIZE : 0;
        A_offs_buff = A_buffer + index * (WARPTILE_M * WARPTILE_K);
        B_offs_buff = B_buffer + index * (WARPTILE_N * WARPTILE_K);

        // Wait for A buffer to be filled
        while (queue[2 * B_LANES * index] != p_state) {
            asm volatile("s_sleep 0");
        }
        // Load A buffer
        #pragma unroll
        for (int op = 0; op < OPS; op++) {
            consumer_smem_to_reg8(A_offs_buff + (op * OP_M * OP_K), reg_A[op]);
        }
        // Mark A buffer as consumed
        queue[2 * B_LANES * index] = p_state + 32;

        // Go through each lanes
        #pragma unroll
        for (int lane = 0; lane < B_LANES; lane++) {

            // Wait for B buffer to be filled
            while (queue[2 * (B_LANES * index + lane) + 1] != p_state) {
                asm volatile("s_sleep 0");
            }
            // Load B buffer
            #pragma unroll
            for (int op = 0; op < OPS; op++) {
                consumer_smem_to_reg16(B_offs_buff + (lane * OP_N * WARPTILE_K) + (op * OP_N * OP_K), reg_B[op]);
            }
            // Mark B buffer as consumed
            queue[2 * (B_LANES * index + lane) + 1] = p_state + 32;

            // Consume registers
            #pragma unroll
            for (int op = 0; op < OPS; op++) {
                reg_D[lane] = __builtin_amdgcn_smfmac_f32_16x16x64_fp8_fp8(
                    reinterpret_cast<fp8_4x2>(reg_A[op]),
                    reinterpret_cast<fp8_4x4>(reg_B[op]),
                    reg_D[lane], 
                    sparsity_indices, // src2
                    7, 1 // cbsz, abid
                );
            }
        }

        // Update index
        index += CONSUMERS;
        p_state = (index >= QSIZE) ? p_state + 64 : p_state;
        b += CONSUMERS;
    }

    // Bring warps back in order
    role_id = b - k_blocks;

    // Fuse complementary registers
    int out_n = 2 * ((thread_id % 16) / 2);
    bool kept;
    #pragma unroll
    for (int i = 0; i < B_LANES; i++) {
        kept = (out_n + i * OP_N) >= dropped_cols;
        reg_D[i][0] = (kept) * (reg_D[i][0] + reg_D[i][1]);
        reg_D[i][1] = (kept) * (reg_D[i][2] + reg_D[i][3]);
    }

    // Relocate on D
    int out_m = (thread_id / 16) * 2 + (thread_id % 2);
    __half2* D_ = reinterpret_cast<__half2*>(D) + (out_m * n + out_n) / 2;

    // Out lane by lane
    int id_to_swap = 1 - threadIdx.x % 2;
    int src_lane = thread_id + 1 - 2 * (thread_id % 2);
    __half2 x;
    for (int i = 0; i < B_LANES; i++) {

        // Swap registers across the warp
        reg_D[i][id_to_swap] = __shfl(
            reg_D[i][id_to_swap], 
            src_lane
        );
        
        // Form the packed f16
        x.x = reg_D[i][0];
        x.y = reg_D[i][1];

        asm volatile(
            "global_atomic_pk_add_f16 %0, %1, off\n\t" : : "v"(&D_[i * OP_N / 2]), "v"(x)
        );
    }
}
