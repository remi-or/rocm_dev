#include "hip/hip_runtime.h"
#include "./consumer.cu"
#include "./producer.cu"

template <typename out_dtype>
void __global__ _tsr_kernel(
    const fp8* __restrict__ A, 
    const fp8* __restrict__ B,
    out_dtype* __restrict__ D, 
    const int m,
    const int n,
    const int k
) {
    // Initialize shared queue
    __shared__ uint8 queue[2 * B_LANES * QSIZE];
    if (threadIdx.x == 0) {
        #pragma unroll
        for (int q = 0; q < 2 * B_LANES * QSIZE; q++) {
            queue[q] = 0;
        }
    }
    // Declare shared buffer
    __shared__ fp8 A_buffer[WARPTILE_M * WARPTILE_K * QSIZE];
    __shared__ fp8 B_buffer[WARPTILE_N * WARPTILE_K * QSIZE];
    __syncthreads();

    // Infer warp specialization
    const int warp_id = threadIdx.x / WARPSIZE;

    // Account for split-k
    A += blockIdx.z * WARPTILE_K * K_BLOCKS(k);
    B += blockIdx.z * WARPTILE_K * K_BLOCKS(k);

    // A producer warp
    if (warp_id < A_PRODUCERS) {
        _tsr_A_producer(A, &A_buffer[0], &queue[0], k); } 
    // B producer warp
    else if (warp_id < A_PRODUCERS + B_LANES * B_PRODUCERS) {
        _tsr_B_producer(B, &B_buffer[0], &queue[0], k); }
    // Consumers warp
    else {
        uint16* q = reinterpret_cast<uint16*>(&queue[0]);
        _tsr_consumer(&A_buffer[0], &B_buffer[0], D, q, n, k);
    }
}

template <typename out_dtype>
void async_gemm(
    const fp8* __restrict__ A, 
    const fp8* __restrict__ B,
    out_dtype* __restrict__ D, 
    const int m, 
    const int n, 
    const int k
) {
    // Check shapes
    if ((m % WARPTILE_M != 0) || (n % WARPTILE_N != 0) || (k % WARPTILE_K != 0)) {
        std::cerr << "Either m, n or k is not divisible by the corresponding WARPTILE_ :";
        std::cerr << m << ", " << n << ", " << k << std::endl;
        exit(1);
    }

    // Prepare kernel launch
    const int grid_m = m / WARPTILE_M;
    const int grid_n = n / WARPTILE_N;
    dim3 grid(grid_m, grid_n, SPLIT_K);

    int warps = 0;
    warps += A_PRODUCERS;
    warps += B_PRODUCERS * B_LANES;
    warps += CONSUMERS * (TIED_CONSUMER ? B_LANES : 1);
    dim3 block(warps * WARPSIZE, 1, 1);

    // Launch kernel
    _tsr_kernel<<<grid, block, 0, 0>>>(A, B, D, m, n, k);
}




// void sparse_k(
//     torch::Tensor& A,
//     torch::Tensor& B,
//     torch::Tensor& D,
//     int64_t W
// ) {
//     const int m = A.size(0);
//     const int n = B.size(1);
//     const int k = A.size(1);
    
//     const fp8* __restrict__ A_ = (const fp8* __restrict__) A.data_ptr(); 
//     const fp8* __restrict__ B_ = (const fp8* __restrict__) B.data_ptr(); 
//     float* __restrict__ D_ = (float* __restrict__) D.data_ptr(); 

//     // Check shapes
//     if ((m % WARPTILE_M != 0) || (n % WARPTILE_N != 0) || (k % WARPTILE_K != 0)) {
//         std::cerr << "Either m, n or k is not divisible by the corresponding WARPTILE_ :";
//         std::cerr << m << ", " << n << ", " << k << std::endl;
//         exit(1);
//     }

    // // Prepare kernel launch
    // const int grid_m = m / WARPTILE_M;
    // const int grid_n = n / WARPTILE_N;
    // dim3 grid(grid_m, grid_n, SPLIT_K);
    
    // int warps = 0;
    // warps += A_PRODUCERS;
    // warps += B_PRODUCERS * B_LANES;
    // warps += CONSUMERS * (TIED_CONSUMER ? B_LANES : 1);
    // dim3 block(warps * WARPSIZE, 1, 1);

//     const at::cuda::OptionalCUDAGuard device_guard(device_of(A));
//     const hipStream_t stream = at::cuda::getCurrentCUDAStream();

//     // Launch kernel
//     _tsr_kernel<float><<<grid, block, 0, stream>>>(A_, B_, D_, m, n, k);
// }
