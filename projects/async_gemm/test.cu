#include "hip/hip_runtime.h"
#include "./../common.cuh"
#include "./src.cu"

#include <hip/hip_fp16.h>

void host_tiled_sum_reduce(
    const fp8* __restrict__ A, 
    const fp8* __restrict__ B,
    float* __restrict__ &D, 
    const int m, 
    const int n, 
    const int k
) {
    float acc;
    const fp8* a;
    const fp8* b;

    // Allocate result tensor
    D = (float*) malloc(m * n * sizeof(float));

    // Square loop
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {

            // Setup K-wise loop
            acc = 0;
            a = A + i * k;
            b = B + j * k;
            // K-wise loop
            for (int l = 0; l < k; l++) {
                acc += __hip_cvt_fp8_to_halfraw(a[l], __HIP_E4M3_FNUZ).data * __hip_cvt_fp8_to_halfraw(b[l], __HIP_E4M3_FNUZ).data;
            }
            // Store back
            D[i * n + j] = acc;
        }
    }
}

int main(int argc, char **argv) {
    HIP_CHECK( hipSetDevice(0) );

    assert(argc==4);
    const int m = atoi(argv[1]);
    const int n = atoi(argv[2]);
    const int k = atoi(argv[3]);

    // Host tensors
    fp8 *hA, *hB;
    float* host_ref;
    fp8 elem = __hip_cvt_float_to_fp8(1.0f, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
    random_host_tensor<fp8>(hA, m * k); // full_host_tensor<fp8>(hA, m * k, elem);
    random_host_tensor<fp8>(hB, n * k); // full_host_tensor<fp8>(hB, n * k, 0); // random_host_tensor<fp8>(hB, n * k);
    // for (int i = 16; i < 32; i++) {
    //     hB[i] = __hip_cvt_float_to_fp8(1.0f, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
    // }
    host_tiled_sum_reduce(hA, hB, host_ref, m, n, k);

    // Device tensors
    fp8 *dA, *dB;
    float* dD;
    tensor_h2d<fp8>(hA, dA, m * k);
    tensor_h2d<fp8>(hB, dB, n * k);
    zero_device_tensor<float>(dD, m * n);

    HIP_CHECK( hipDeviceSynchronize() );
    async_gemm(dA, dB, dD, m, n, k);
    HIP_CHECK( hipDeviceSynchronize() );

    // Transfer result and free device tensors
    float* host_result;
    tensor_d2h<float>(dD, host_result, m * n);
    HIP_CHECK( hipDeviceSynchronize() );
    HIP_CHECK(hipFree(dA));
    HIP_CHECK(hipFree(dB));
    HIP_CHECK(hipFree(dD));

    // Compare the two host tensors
    float delta;
    float sum_delta = 0.0f;
    float max_delta = abs(host_result[0] - host_ref[0]);
    for (int k = 0; k < m * n; k++) {
        delta = abs(host_result[k] - host_ref[k]);
        sum_delta += delta;
        max_delta = (delta > max_delta) ? delta : max_delta;
        // std::cout << host_ref[k] << ":" << host_result[k] << ", ";
    }
    std::cout << "{\"max_delta\": " << max_delta << ", \"total_delta\": " << sum_delta << "}";

    // Free host-side tensors
    free(hA);
    free(hB);
    free(host_ref);
    free(host_result);
}
