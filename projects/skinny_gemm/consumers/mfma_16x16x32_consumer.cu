#include "hip/hip_runtime.h"
template<int A_LANES, int B_LANES, int QSIZE, int OPS>
void __device__ consume_tiles_dense_16x16x32(
    fp8* A_buffer,
    fp8* B_buffer,
    half* D,
    float scale,
    const int consumers,
    int* a_queue,
    int &index,
    int &p_state,
    int &role_id,
    const int dropped_rows,
    const int dropped_cols,
    const int n,
    const int k,
    const int k_blocks
) {
    // Compile-time constants
    static constexpr int E_PER_BANK = 4;
    static constexpr int OP_M = 16;
    static constexpr int OP_N = 16;
    static constexpr int OP_K = 32;
    static constexpr int WARPTILE_M = OP_M * A_LANES;
    static constexpr int WARPTILE_N = OP_N * B_LANES;
    static constexpr int WARPTILE_K = OP_K * OPS;

    // Compute thread position
    const int lane_id = get_lane_id();

    // Relocate in buffers
    A_buffer += (lane_id % 16) * E_PER_BANK + (lane_id / 32) * 16 * E_PER_BANK;
    A_buffer += (lane_id % 32 >= 16) ? NB_BANKS * E_PER_BANK * 2 : 0;

    B_buffer += (lane_id % 16) * E_PER_BANK + (lane_id / 32) * 16 * E_PER_BANK;
    B_buffer += (lane_id % 32 >= 16) ? NB_BANKS * E_PER_BANK * 2 : 0;

    // Declare input registers
    fp8x8 reg_A[A_LANES][OPS];
    fp8x8 reg_B[OPS];

    // Initialize output registers
    f32x4 reg_D[A_LANES][B_LANES];
    #pragma unroll
    for (int a_lane = 0; a_lane < A_LANES; a_lane++) {
        #pragma unroll
        for (int b_lane = 0; b_lane < B_LANES; b_lane++) {
            reg_D[a_lane][b_lane][0] = 0.0f;
            reg_D[a_lane][b_lane][1] = 0.0f;
            reg_D[a_lane][b_lane][2] = 0.0f;
            reg_D[a_lane][b_lane][3] = 0.0f;
        }
    }

    // K-wise loop
    int b = role_id;
    int* b_queue = a_queue + A_LANES * QSIZE;

    while (b < k_blocks) {

        // Account for cyclic queue
        index -= (index >= QSIZE) ? QSIZE : 0;
        fp8* A_offs_buff = A_buffer + index * (WARPTILE_M * WARPTILE_K);
        fp8* B_offs_buff = B_buffer + index * (WARPTILE_N * WARPTILE_K);

        // Go through all B lanes
        #pragma unroll
        for (int b_lane = 0; b_lane < B_LANES; b_lane++) {

            // Wait for B buffer to be filled
            while (b_queue[B_LANES * index + b_lane] != p_state) {
                asm volatile("s_sleep 0");
            }
            // Load B buffer
            #pragma unroll
            for (int op = 0; op < OPS; op++) {
                consumer_smem_to_reg(B_offs_buff + (b_lane * OP_N * WARPTILE_K) + (op * OP_N * OP_K), reg_B[op]);
            }
            // Mark B buffer as consumed (wait for last B lane)
            if (b_lane == B_LANES - 1) {
                asm volatile("s_waitcnt lgkmcnt(0)");
            }
            b_queue[B_LANES * index + b_lane] = p_state + 1;

            // Go through all a lanes
            #pragma unroll
            for (int a_lane = 0; a_lane < A_LANES; a_lane++) {

                // If this is the first B lane, fill A registers
                if (b_lane == 0) {
                    // Wait for A buffer to be filled
                    while (a_queue[A_LANES * index + a_lane] != p_state) {
                        asm volatile("s_sleep 0");
                    }
                     // Load A buffer
                    #pragma unroll
                    for (int op = 0; op < OPS; op++) {
                        consumer_smem_to_reg(A_offs_buff + (a_lane * OP_M * WARPTILE_K) + (op * OP_M * OP_K), reg_A[a_lane][op]);
                    }
                    // Mark A buffer as consumed (wait only if there is not other B lane than B lane 0)
                    // NOTE: not sure this is needed, but then again who uses B_lanes == 1
                    if constexpr (B_LANES == 1) {
                        asm volatile("s_waitcnt lgkmcnt(0)");
                    }
                    a_queue[A_LANES * index + a_lane] = p_state + 1;
                }

                // Consume registers
                #pragma unroll
                for (int op = 0; op < OPS; op++) {
                    reg_D[a_lane][b_lane] = __builtin_amdgcn_mfma_f32_16x16x32_fp8_fp8(
                        reinterpret_cast<long>(reg_A[a_lane][op]),
                        reinterpret_cast<long>(reg_B[op]),
                        reg_D[a_lane][b_lane],
                        0, // src2
                        0, // cbsz
                        0  // abid
                    );
                }
            }
        }
        // Update index
        index += consumers;
        p_state = (index >= QSIZE) ? (p_state + 2) : p_state;
        b += consumers;
    }

    // Bring warps back in order
    role_id = b - k_blocks;

    // Infer the current column in D
    int out_n = 2 * ((lane_id % 16) / 2);

    // Prepare swapping variables
    int id_to_swap = 1 - lane_id % 2;
    int src_lane = lane_id + 1 - 2 * (lane_id % 2);

    // Loop over all a lanes
    #pragma unroll
    for (int a_lane = 0; a_lane < A_LANES; a_lane++) {

        // Loop over all b lanes
        #pragma unroll
        for (int b_lane = 0; b_lane < B_LANES; b_lane++) {

            // Scaling
            float final_scale = (out_n + b_lane * OP_N) >= dropped_cols ? scale : 0.0f;
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                reg_D[a_lane][b_lane][j] *= final_scale;
            }

            // Swapping
            #pragma unroll
            for (int j = 0; j < 4; j+=2) {
                reg_D[a_lane][b_lane][id_to_swap + j] = __shfl(reg_D[a_lane][b_lane][id_to_swap + j], src_lane);
            }
        }

        // Infer the starting row in D
        int out_m0 = a_lane * OP_M + (lane_id / 16) * 4 + (lane_id % 2);
        __half2* D_packed = reinterpret_cast<__half2*>(D + (out_m0 * n + out_n));

        // Loop over all rows
        #pragma unroll
        for (int row = 0; row < 2; row++) {

            // Quit if we are in dropped rows territory
            if (out_m0 + dropped_rows >= WARPTILE_M - 2 * row) { return; }

            // Out lane by lane
            __half2 x;
            #pragma unroll
            for (int b_lane = 0; b_lane < B_LANES; b_lane++) {
                x.x = __float2half(reg_D[a_lane][b_lane][2 * row]); // TODO: packed conversion
                x.y = __float2half(reg_D[a_lane][b_lane][2 * row + 1]);
                asm volatile("global_atomic_pk_add_f16 %0, %1, off\n\t" : : "v"(&D_packed[b_lane * (OP_N / 2)]), "v"(x));
            }

            // Advance two rows (one in packed)
            D_packed += n;
        }
    }

    // Debug: input register look-up
    // if (threadIdx.x % WARPSIZE == 0) {
    //     for (int a_lane = 0; a_lane < A_LANES; a_lane++) {
    //         for (int op = 0; op < OPS; op++) {
    //             printf("Op %d: ", op);
    //             for (int i = 0; i < 16; i++) {
    //                 printf("%f, ", (float)__hip_cvt_fp8_to_halfraw(reg_A[a_lane][op][i], __HIP_E4M3_FNUZ).data);
    //             }
    //             printf("\n");
    //         }
    //     }
    // }

    // Debug: bank look-up
    // if (threadIdx.x % WARPSIZE == 0) {
    //     for (int bank = 0; bank < NB_BANKS; bank++) {
    //         for (int line = 0; line < 16; line++) {
    //             printf("B %d / L %d: ", bank, line);
    //             for (int elem = 0; elem < E_PER_BANK; elem++) {
    //                 int index = bank * (E_PER_BANK * NB_BANKS) + line * E_PER_BANK + elem;
    //                 float x = __hip_cvt_fp8_to_halfraw(A_buffer[index], __HIP_E4M3_FNUZ).data;
    //                 printf("%f, ", x);
    //             }
    //             printf("\n");
    //         }
    //     }
    //     printf("\n");
    // }
}
